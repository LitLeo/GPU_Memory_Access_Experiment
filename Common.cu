#include "hip/hip_runtime.h"
#include "Common.h"

string EnumToString(enum data_form df)
{
    switch (df)
    {
    case df_1D:
        return "df_1D";
    case df_2D:
        return "df_2D";
    case df_tree:
        return "df_tree";
    }
    return "";
}
string EnumToString(enum access_mode am)
{
    switch (am)
    {
    case am_sequential:
        return "am_sequential";
    case am_step:
        return "am_step";
    case am_random:
        return "am_random";
    case am_standard_normal:
        return "am_standard_normal";
    case am_poisson:
        return "am_poisson";
    case am_geometric:
        return "am_geometric";
    case am_exponential:
        return "am_exponential";
    }
    return "";
}
string EnumToString(enum data_content dc)
{
    switch (dc)
    {
    case dc_random:
        return "dc_random";
    case dc_standard_normal:
        return "dc_standard_normal";
    case dc_poisson:
        return "dc_poisson";
    case dc_uniform:
        return "dc_uniform";
    case dc_geometric:
        return "dc_geometric";
    case dc_exponential:
        return "dc_exponential";
    }
    return "";
}

void print(DATA_TYPE* data_1D, int size)
{
    for (int i = 0; i < size; i++)
    {
        cout << (float)data_1D[i] << " ";
    }
    cout << endl;
}

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
        C[i] = A[i] + B[i];
}

// warmup 函数，用于计时时 warmup GPU，实际是一个 vector 相加
void warmup()
{
    int numElements = 1024;
    size_t size = numElements * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);

    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

}

