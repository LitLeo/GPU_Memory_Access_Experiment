#include "hip/hip_runtime.h"
#include "Distribution.h"

#include <stdlib.h>
#include <time.h>

// 代码实现参考
// http://blog.csdn.net/zhengnanlee/article/details/12619483
// http://m.blog.csdn.net/blog/asiaLIYAZHOU/45509047
// http://www.cnblogs.com/yeahgis/archive/2012/07/15/2592696.html

// 随机函数，私有函数，供该文件内其他函数调用
double  _random(double min, double max)
{
    int nbRand = rand() % 10001;
    return (min + nbRand*(max - min) / 10000);
}

// 随机分布函数
int random(int min, int max, int size, DATA_TYPE* data)
{
    if (data == NULL)
        return -1;
    srand((unsigned)time(NULL));
    for (int i = 0; i < size; i++)
        data[i] = _random(min, max);
    return 0;
}
// 随机分布函数，用于生成访问下标
int random(int min, int max, int* data)
{
    if (data == NULL)
        return -1;
    srand((unsigned)time(NULL));
    for (int i = 0; i < max; i++)
        data[i] = _random(min, max);
    return 0;
}
/*为树节点初始化数据*/
int random(int min, int max, int size, Node *nodes)
{
    if (nodes == NULL)
        return -1;
    srand((unsigned)time(NULL));
    for (int i = 0; i < size; i++)
        nodes[i].data = _random(min, max);
    return 0;
}

// 标准正态分布函数
double Normal(double x, double miu, double sigma)
{
    return 1.0 / sqrt(2 * PI*sigma) * exp(-1 * (x - miu)*(x - miu) / (2 * sigma*sigma));
}
double NormalRandom(double miu, double sigma, double min, double max)
{
    double x;
    double dScope;
    double y;
    do
    {
        x = _random(min, max);
        y = Normal(x, miu, sigma);
        dScope = _random(0, Normal(miu, miu, sigma));
    } while (dScope > y);
    return x;
}
int standard_normal(int min, int max, int miu, int sigma, int size, DATA_TYPE* data)
{
    if (data == NULL || max <= min)
        return -1;
    srand((unsigned)time(NULL));
    for (int i = 0; i < size; i++)
    {
        data[i] = NormalRandom(miu, sigma, min, max);
    }
    return 0;
}
// 标准正态分布函数，用于生成访问下标
int standard_normal(int min, int max, int miu, int sigma, int* data)
{
    if (data == NULL || max <= min)
        return -1;
    srand((unsigned)time(NULL));
    for (int i = 0; i < max; i++)
    {
        data[i] = NormalRandom(miu, sigma, min, max);
    }
    return 0;
}
/*为树节点初始化数据*/
int standard_normal(int min, int max, int miu, int sigma, int size, Node *nodes)
{
    if (nodes == NULL || max <= min)
        return -1;
    srand((unsigned)time(NULL));
    for (int i = 0; i < size; i++)
    {
        nodes[i].data = NormalRandom(miu, sigma, min, max);
    }
    return 0;
}

// 泊松分布
int _possion(int Lambda)
{
    int  k = 0;
    long double p = 1.0;
    long double l = exp(-Lambda);
    while(p>=l)
    {
        double u = (float)(rand() %100) / 100;
        p *= u;
        k++;
    }
    return k-1;
}
int poisson(int min, int max, int Lambda, int size, DATA_TYPE* data)
{
    if (data == NULL || max <= min)
        return -1;
    srand((unsigned)time(NULL));

    for (int i = 0; i < size; i++)
    {
        data[i] = _possion(Lambda) % (max - min);
    }
    return 0;
}
// 泊松分布函数，用于生成访问下标
int poisson(int min, int max, int Lambda, int* data)
{
    if (data == NULL || max <= min)
        return -1;
    srand((unsigned)time(NULL));

    for (int i = 0; i < max; i++)
    {
        data[i] = _possion(Lambda) % (max - min);
    }
    return 0;
}
/*为树节点初始化数据*/
int poisson(int min, int max, int Lambda, int size, Node *nodes)
{
    if (nodes == NULL || max <= min)
        return -1;
    srand((unsigned)time(NULL));

    for (int i = 0; i < size; i++)
    {
        nodes[i].data = _possion(Lambda) % (max - min);
    }
    return 0;
}

// 均匀分布
int uniform(int min, int max, int size, DATA_TYPE* data)
{
    if (data == NULL || max <= min)
        return -1;

    for (int i = 0; i < size; i++)
        data[i] = (min + i) % (max - min);
    return 0;
}
// 均匀分布函数，用于生成访问下标
int uniform(int min, int max, int* data)
{
    if (data == NULL || max <= min)
        return -1;

    for (int i = 0; i < max; i++)
        data[i] = (min + i) % (max - min);
    return 0;
}
/*为树节点初始化数据*/
int uniform(int min, int max, int size, Node *nodes)
{
    if (nodes == NULL || max <= min)
        return -1;

    for (int i = 0; i < size; i++)
        nodes[i].data = (min + i) % (max - min);
    return 0;
}

// 几何分布
long randomGeometric(double  probability)
{
    long rnd = 0;
    while(true)
    {
        rnd++;
        double pV = (double)rand()/(double)RAND_MAX;
        if (pV<probability)
        {
            break;
        }
    }
    return rnd;
}
int geometric(double probability, int min, int max, int size, DATA_TYPE* data)
{
    if (data == NULL || max <= min)
        return -1;
     srand((unsigned)time(NULL));
     for (int i = 0; i < size; i++)
     {
         data[i] = randomGeometric(probability) % (max - min);
     }
     return 0;
}
// 几何分布函数，用于生成访问下标
int geometric(double probability, int min, int max, int* data)
{
    if (data == NULL || max <= min)
        return -1;
     srand((unsigned)time(NULL));
     for (int i = 0; i < max; i++)
     {
         data[i] = randomGeometric(probability) % (max - min);
     }
     return 0;
}
/*为树节点初始化数据*/
int geometric(double probability, int min, int max, int size, Node *nodes)
{
    if (nodes == NULL || max <= min)
        return -1;
     srand((unsigned)time(NULL));
     for (int i = 0; i < size; i++)
     {
         nodes[i].data = randomGeometric(probability) % (max - min);
     }
     return 0;
}

// 指数分布
double randomExponential(double lambda)
{
    double pv = 0.0;
    pv = (double)(rand()%100)/100;
    while(pv == 0)
    {
        pv = (double)(rand() % 100)/100;
    }
    pv = (-1  / lambda)*log(1-pv);
    return pv;
}
int exponential(double lambda, int min, int max, int size, DATA_TYPE* data)
{
    if (data == NULL || max <= min)
       return -1;
    srand((unsigned)time(NULL));

    for (int i = 0; i < size; i++)
        data[i] = (int)randomExponential(lambda) % (max - min);
    return 0;
}
// 指数分布函数，用于生成访问下标
int exponential(double lambda, int min, int max, int* data)
{
    if (data == NULL || max <= min)
       return -1;
    srand((unsigned)time(NULL));

    for (int i = 0; i < max; i++)
        data[i] = (int)randomExponential(lambda) % (max - min);
    return 0;
}
/*为树节点初始化数据*/
int exponential(double lambda, int min, int max, int size, Node *nodes)
{
    if (nodes == NULL || max <= min)
        return -1;
    srand((unsigned)time(NULL));

    for (int i = 0; i < size; i++)
        nodes[i].data = (int)randomExponential(lambda) % (max - min);
    return 0;
}

// device 端函数，用于在核函数中根据不同的访问方式产生访问下标
// __device__ __host__ int dev_sequential(int index)
// {
//     return index;
// }
// __device__ __host__ int dev_step(int index, int step)
// {
//     return index * step;
// }
// __device__ __host__ int dev_random()
// {

// }
// __device__ __host__ int dev_standard_normal(int index)
// {

// }
// __device__ __host__ int dev_poisson(int index)
// {

// }
// __device__ __host__ int dev_geometric(int index)
// {

// }
// __device__ __host__ int dev_exponential(int index)
// {

// }
