#include <iostream>
#include "Case.h"
using namespace std;
#ifndef LOOP
#define LOOP 1  // 因为使用了warmup函数，所以就不需要循环多次了
#endif
int main()
{
    // global 测试数据大小
    const int global_size_num = 5; // 数组大小;
    const int global_size[global_size_num] = {512, 1024, 4096, 10240, 40960};

    // constant 和 
    // const int con_size_num = 4;
    // const int con_size[con_size_num] = {512, 1024, 4096, 10240};
    // shared 测试数据大小,这里的大小设置受 DATA_TYPE 的影响。
    const int shared_size_num = 4;
    const int shared_size[shared_size_num] = {512, 1024, 4096, 10240};
    // block size
    const int block_size_num = 3;
    const int block_size[block_size_num] = {256, 512, 1024};
    // 六种数据内部分布，具体见common.h
    const int dc = 6;
    // 七种访问数据类型，具体见common.h
    const int am = 7;
    // 每个线程访问多少数据
    const int am_num_num = 3;
    const int am_num[am_num_num] = {1, 2, 4};
    
    const int step_num = 3;
    const int step[step_num] = {1, 2, 4};

    // 1D Global
    for (int gs = 0; gs < global_size_num; gs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_1D;
                c.size = global_size[gs];
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();

                for (int bs= 0; bs < block_size_num; bs++)
                    for (int an = 0; an < am_num_num; an++) {   
                        c.thread_num = c.size;         // 线程数与数据量大小相同
                        c.block_size = block_size[bs];   
                        c.am_num = am_num[an];

                        cout<<endl<<(c.df)
                        << " Global:"
                        <<" size="<< c.size 
                        <<" access_num_per_thread=" << am_num[an]
                        <<" block_size="<<c.block_size
                        <<" data_content="<< (c.dc)
                        <<" access_mode="<<(c.am);

                        float runTime;

                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0) {
                                    cout<<endl<<(c.df)
                                    << " Global:"
                                    <<" size="<< c.size 
                                    <<" access_num_per_thread=" << am_num[an]
                                    <<" block_size="<<c.block_size
                                    <<" data_content="<< (c.dc)
                                    <<" access_mode="<<(c.am);
                                }

                                cout << " step=" << step[s];

                                // warmup();
                                // hipEvent_t start, stop;
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.global_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {

                            // warmup();
                            // hipEvent_t start, stop;
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.global_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }

                    }
            }

    // 1D constant
    // for (int cs = 0; cs < con_size_num; cs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_1D;
                c.size = 512;//  这里每一次都需要手动设置
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();

                for (int bs= 0; bs < block_size_num; bs++)
                    for (int an = 0; an < am_num_num; an++) {   
                        c.thread_num = c.size;         // 线程数与数据量大小相同
                        c.block_size = block_size[bs];   
                        c.am_num = am_num[an];

                        cout<<endl<<(c.df)
                        << " Constant:"
                        <<" size="<< c.size 
                        <<" access_num_per_thread=" << am_num[an]
                        <<" block_size="<<c.block_size
                        <<" data_content="<< (c.dc)
                        <<" access_mode="<<(c.am);

                        float runTime;
                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0) {
                                    cout<<endl<<(c.df)
                                    << " Constant:"
                                    <<" size="<< c.size 
                                    <<" access_num_per_thread=" << am_num[an]
                                    <<" block_size="<<c.block_size
                                    <<" data_content="<< (c.dc)
                                    <<" access_mode="<<(c.am);
                                }

                                cout << " step=" << step[s];
                                
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.constant_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.constant_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }
    // 1D shared
    for (int ss = 0; ss < shared_size_num; ss++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_1D;
                c.size = shared_size[ss];
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();

                for (int bs= 0; bs < block_size_num; bs++)
                    for (int an = 0; an < am_num_num; an++) {   
                        c.thread_num = c.size;         // 线程数与数据量大小相同
                        c.block_size = block_size[bs];   
                        c.am_num = am_num[an];

                        cout<<endl<<(c.df)
                        << " Shared:"
                        <<" size="<< c.size 
                        <<" access_num_per_thread=" << am_num[an]
                        <<" block_size="<<c.block_size
                        <<" data_content="<< (c.dc)
                        <<" access_mode="<<(c.am);

                        float runTime;
                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    cout<<endl<<(c.df)
                                    << " Shared:"
                                    <<" size="<< c.size 
                                    <<" access_num_per_thread=" << am_num[an]
                                    <<" block_size="<<c.block_size
                                    <<" data_content="<< (c.dc)
                                    <<" access_mode="<<(c.am);
                                }

                                cout << " step=" << step[s];

                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.shared_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.shared_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }

    const int col_num = 2;
    const int col[col_num] = {256, 512};
    // 二维数组分别在global、constant、shared中进行访存
    // 2D global
    for (int gs = 0; gs < global_size_num; gs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++)
                for (int cn = 0; cn < col_num; cn++) {
                    Case c;
                    c.df = df_2D;
                    c.size = global_size[gs];
                    c.c = col[cn];  
                    c.r = (c.size + c.c - 1) / c.c;
                    
                    c.dc = (enum data_content)_dc;
                    c.am = (enum access_mode)_am;
                    // c.initData();
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    for (int an = 0; an < am_num_num; an++)
                        for (int bs= 0; bs < block_size_num; bs++) {
                            c.block_size = block_size[bs];
                            c.am_num = am_num[an];

                            cout<<endl<<(c.df)
                            << " Global:"
                            <<" size="<< c.size 
                            <<" r=" << c.r
                            <<",c=" << c.c
                            <<" access_num_per_thread=" << am_num[an]
                            <<" block_size="<<c.block_size
                            <<" data_content="<< (c.dc)
                            <<" access_mode="<<(c.am);
  
                            float runTime;
                            // warmup();
                            // hipEvent_t start, stop;
                            // step access mode 
                            if (_am == 1) {
                                for (int s = 0; s < step_num; ++s) {
                                    c.step = step[s];

                                    if (s != 0)
                                    {
                                        cout<<endl<<(c.df)
                                        << " Global:"
                                        <<" size="<< c.size 
                                        <<" r=" << c.r
                                        <<",c=" << c.c
                                        <<" access_num_per_thread=" << am_num[an]
                                        <<" block_size="<<c.block_size
                                        <<" data_content="<< (c.dc)
                                        <<" access_mode="<<(c.am);
                                    }

                                    cout << " step=" << step[s];

                                    // hipEventCreate(&start);
                                    // hipEventCreate(&stop);
                                    // hipEventRecord(start, 0);
                                    // for (int i = 0; i < LOOP; i++) 
                                        // c.global_run();
                                    // hipEventRecord(stop, 0);
                                    // hipEventSynchronize(stop);
                                    // hipEventElapsedTime(&runTime, start, stop);
                                    // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                                }
                            } else {
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.global_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        }
                }
    
    // 2D constant
    // for (int cs = 0; cs < con_size_num; cs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++)
                for (int cn = 0; cn < col_num; cn++) {
                    Case c;
                    c.df = df_2D;
                    c.size = 512;//con_size[cs];
                    c.c = col[cn];  
                    c.r = (c.size + c.c - 1) / c.c;
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    c.dc = (enum data_content)_dc;
                    c.am = (enum access_mode)_am;
                    // c.initData();
                    for (int an = 0; an < am_num_num; an++)
                        for (int bs= 0; bs < block_size_num; bs++)
                        {
                            c.am_num = am_num[an];
                            c.block_size = block_size[bs];

                            cout<<endl<<(c.df)
                            << " Constant:"
                            <<" size="<< c.size 
                            <<" access_num_per_thread=" << am_num[an]
                            <<" block_size="<<c.block_size
                            <<" data_content="<< (c.dc)
                            <<" access_mode="<<(c.am);

                            float runTime;
                            // warmup();
                            // hipEvent_t start, stop;
                            // step access mode 
                            if (_am == 1) {
                                for (int s = 0; s < step_num; ++s) {
                                    c.step = step[s];

                                    if (s != 0)
                                    {
                                        cout<<endl<<(c.df)
                                        << " Constant:"
                                        <<" size="<< c.size 
                                        <<" access_num_per_thread=" << am_num[an]
                                        <<" block_size="<<c.block_size
                                        <<" data_content="<< (c.dc)
                                        <<" access_mode="<<(c.am);
                                    }

                                    cout << " step=" << step[s];

                                    // hipEventCreate(&start);
                                    // hipEventCreate(&stop);
                                    // hipEventRecord(start, 0);
                                    // for (int i = 0; i < LOOP; i++) 
                                        // c.constant_run();
                                    // hipEventRecord(stop, 0);
                                    // hipEventSynchronize(stop);
                                    // hipEventElapsedTime(&runTime, start, stop);
                                    // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                                }
                            } else {
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.constant_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        }
                }
   
    // 2D shared
    for (int ss = 0; ss < shared_size_num; ss++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++)
                for (int cn = 0; cn < col_num; cn++) {
                    Case c;
                    c.df = df_2D;
                    c.size = shared_size[ss];
                    c.c = col[cn];  
                    c.r = (c.size + c.c - 1) / c.c;
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    c.dc = (enum data_content)_dc;
                    c.am = (enum access_mode)_am;
                    // c.initData();
                    for (int an = 0; an < am_num_num; an++)
                        for (int bs= 1; bs < block_size_num; bs++) {
                            c.block_size = block_size[bs];
                            c.am_num = am_num[an];

                            cout<<endl<<(c.df)
                            << " Shared:"
                            <<" size="<< c.size 
                            <<" access_num_per_thread=" << am_num[an]
                            <<" block_size="<<c.block_size
                            <<" data_content="<< (c.dc)
                            <<" access_mode="<<(c.am);
                            
                            float runTime;
                            // warmup();
                            // hipEvent_t start, stop;
                            // step access mode 
                            if (_am == 1) {
                                for (int s = 0; s < step_num; ++s) {
                                    c.step = step[s];

                                    if (s != 0)
                                    {
                                        cout<<endl<<(c.df)
                                        << " Shared:"
                                        <<" size="<< c.size 
                                        <<" access_num_per_thread=" << am_num[an]
                                        <<" block_size="<<c.block_size
                                        <<" data_content="<< (c.dc)
                                        <<" access_mode="<<(c.am);
                                    }

                                    cout << " step=" << step[s];

                                    // hipEventCreate(&start);
                                    // hipEventCreate(&stop);
                                    // hipEventRecord(start, 0);
                                    // for (int i = 0; i < LOOP; i++) 
                                        // c.shared_run();
                                    // hipEventRecord(stop, 0);
                                    // hipEventSynchronize(stop);
                                    // hipEventElapsedTime(&runTime, start, stop);
                                    // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                                }
                            } else {
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.shared_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        }
                }
    
    // Tree Global
    for (int gs = 0; gs < global_size_num; gs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_tree;
                c.size = global_size[gs];
                c.thread_num = c.size;         // 线程数与数据量大小相同
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();
                for (int an = 0; an < am_num_num; an++) 
                    for (int bs= 0; bs < block_size_num; bs++){
                        c.am_num = am_num[an];
                        c.block_size = block_size[bs];

                        cout<<endl<<(c.df)
                        << " Global:"
                        <<" size="<< c.size 
                        <<" access_num_per_thread=" << am_num[an]
                        <<" block_size="<<c.block_size
                        <<" data_content="<< (c.dc)
                        <<" access_mode="<<(c.am);

                        float runTime;

                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    cout<<endl<<(c.df)
                                    << " Global:"
                                    <<" size="<< c.size 
                                    <<" access_num_per_thread=" << am_num[an]
                                    <<" block_size="<<c.block_size
                                    <<" data_content="<< (c.dc)
                                    <<" access_mode="<<(c.am);
                                }

                                cout << " step=" << step[s];

                                // warmup();
                                // hipEvent_t start, stop;
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.global_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {

                            // warmup();
                            // hipEvent_t start, stop;
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.global_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }
    // Tree constant
    // for (int cs = 0; cs < con_size_num; cs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_tree;
                c.size = 512;//  这里每一次都需要手动设置
                c.thread_num = c.size; // 线程数与数据量大小相同
                
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();
                for (int an = 0; an < am_num_num; an++)
                    for (int bs= 0; bs < block_size_num; bs++) {
                        c.am_num = am_num[an];
                        c.block_size = block_size[bs];

                        cout<<endl<<(c.df)
                        << " Constant:"
                        <<" size="<< c.size 
                        <<" access_num_per_thread=" << am_num[an]
                        <<" block_size="<<c.block_size
                        <<" data_content="<< (c.dc)
                        <<" access_mode="<<(c.am);

                        float runTime;
                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    cout<<endl<<(c.df)
                                    << " Constant:"
                                    <<" size="<< c.size 
                                    <<" access_num_per_thread=" << am_num[an]
                                    <<" block_size="<<c.block_size
                                    <<" data_content="<< (c.dc)
                                    <<" access_mode="<<(c.am);
                                }

                                cout << " step=" << step[s];
                                
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.constant_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.constant_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }
    
    // Tree shared
    // 在这里要重新设计大小，因为sizeof(tree) = 4+4+sizof(DATA_TYPE)，
    // 同时还要考虑结构体对齐问题
    // DATA_TYPE = unsigned char时, sizeof(tree) = 12B，
    const int tree_shared_size_num = 4;
    const int tree_shared_size[tree_shared_size_num] = {256, 512, 1024, 2048};

    for (int tss = 0; tss < tree_shared_size_num; tss++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_tree;
                c.size = tree_shared_size[tss];
                c.thread_num = c.size;         // 线程数与数据量大小相同
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();
                for (int an = 0; an < am_num_num; an++) 
                    for (int bs= 0; bs < block_size_num; bs++){
                        c.am_num = am_num[an];
                        c.block_size = block_size[bs];

                        cout<<endl<<(c.df)
                        << " Shared:"
                        <<" size="<< c.size 
                        <<" access_num_per_thread=" << am_num[an]
                        <<" block_size="<<c.block_size
                        <<" data_content="<< (c.dc)
                        <<" access_mode="<<(c.am);

                        float runTime;
                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    cout<<endl<<(c.df)
                                    << " Shared:"
                                    <<" size="<< c.size 
                                    <<" access_num_per_thread=" << am_num[an]
                                    <<" block_size="<<c.block_size
                                    <<" data_content="<< (c.dc)
                                    <<" access_mode="<<(c.am);
                                }

                                cout << " step=" << step[s];

                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.shared_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.shared_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // cout << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }


    return 0;
}


