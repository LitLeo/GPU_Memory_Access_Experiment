#include "GenerateParams.h"

void generate_params()
{
    char* filename = "params.txt";
    remove(filename);
    ofstream outfile("params.txt");

    // global 测试数据大小
    const int global_size_num = 5; // 数组大小;
    const int global_size[global_size_num] = {1024, 4096, 10240, 40960, 102400};

    // constant 和 
    // const int con_size_num = 4;
    // const int con_size[con_size_num] = {512, 1024, 4096, 10240};
    // shared 测试数据大小,这里的大小设置受 DATA_TYPE 的影响。
    const int shared_size_num = 4;
    const int shared_size[shared_size_num] = {512, 1024, 4096, 10240};
    // block size
    const int block_size_num = 3;
    const int block_size[block_size_num] = {256, 512, 1024};
    // 六种数据内部分布，具体见common.h
    const int dc = 6;
    // 七种访问数据类型，具体见common.h
    const int am = 7;
    // 每个线程访问多少数据
    const int am_num_num = 3;
    const int am_num[am_num_num] = {1, 2, 4};
    
    const int step_num = 3;
    const int step[step_num] = {1, 2, 4};

    const int col_num = 2;
    const int col[col_num] = {256, 512};
/*
    // 1D Global
    for (int gs = 0; gs < global_size_num; gs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_1D;
                c.size = global_size[gs];
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();

                for (int bs= 0; bs < block_size_num; bs++)
                    for (int an = 0; an < am_num_num; an++) {   
                        c.thread_num = c.size;         // 线程数与数据量大小相同
                        c.block_size = block_size[bs];   
                        c.am_num = am_num[an];

                        outfile<<endl<<(c.df)
                        << " Global:"
                        <<" "<< c.size 
                        <<" " << am_num[an]
                        <<" "<<c.block_size
                        <<" "<< (c.dc)
                        <<" "<<(c.am);


                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0) {
                                    outfile<<endl<<(c.df)
                                    << " Global:"
                                    <<" "<< c.size 
                                    <<" " << am_num[an]
                                    <<" "<<c.block_size
                                    <<" "<< (c.dc)
                                    <<" "<<(c.am);
                                }

                                outfile << " " << step[s];

                                // warmup();
                                // hipEvent_t start, stop;
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.global_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {

                            // warmup();
                            // hipEvent_t start, stop;
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.global_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }

                    }
            }

    // 1D constant
    // for (int cs = 0; cs < con_size_num; cs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_1D;
                c.size = 512;//  这里每一次都需要手动设置
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();

                for (int bs= 0; bs < block_size_num; bs++)
                    for (int an = 0; an < am_num_num; an++) {   
                        c.thread_num = c.size;         // 线程数与数据量大小相同
                        c.block_size = block_size[bs];   
                        c.am_num = am_num[an];

                        outfile<<endl<<(c.df)
                        << " Constant:"
                        <<" "<< c.size 
                        <<" " << am_num[an]
                        <<" "<<c.block_size
                        <<" "<< (c.dc)
                        <<" "<<(c.am);

                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0) {
                                    outfile<<endl<<(c.df)
                                    << " Constant:"
                                    <<" "<< c.size 
                                    <<" " << am_num[an]
                                    <<" "<<c.block_size
                                    <<" "<< (c.dc)
                                    <<" "<<(c.am);
                                }

                                outfile << " " << step[s];
                                
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.constant_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.constant_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }
    // 1D shared
    for (int ss = 0; ss < shared_size_num; ss++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_1D;
                c.size = shared_size[ss];
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();

                for (int bs= 0; bs < block_size_num; bs++)
                    for (int an = 0; an < am_num_num; an++) {   
                        c.thread_num = c.size;         // 线程数与数据量大小相同
                        c.block_size = block_size[bs];   
                        c.am_num = am_num[an];

                        outfile<<endl<<(c.df)
                        << " Shared:"
                        <<" "<< c.size 
                        <<" " << am_num[an]
                        <<" "<<c.block_size
                        <<" "<< (c.dc)
                        <<" "<<(c.am);

                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    outfile<<endl<<(c.df)
                                    << " Shared:"
                                    <<" "<< c.size 
                                    <<" " << am_num[an]
                                    <<" "<<c.block_size
                                    <<" "<< (c.dc)
                                    <<" "<<(c.am);
                                }

                                outfile << " " << step[s];

                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.shared_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.shared_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }

    // 二维数组分别在global、constant、shared中进行访存
    // 2D global
    for (int gs = 0; gs < global_size_num; gs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++)
                for (int cn = 0; cn < col_num; cn++) {
                    Case c;
                    c.df = df_2D;
                    c.size = global_size[gs];
                    c.c = col[cn];  
                    c.r = (c.size + c.c - 1) / c.c;
                    
                    c.dc = (enum data_content)_dc;
                    c.am = (enum access_mode)_am;
                    // c.initData();
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    for (int an = 0; an < am_num_num; an++)
                        for (int bs= 0; bs < block_size_num; bs++) {
                            c.block_size = block_size[bs];
                            c.am_num = am_num[an];

                            outfile<<endl<<(c.df)
                            << " Global:"
                            <<" "<< c.size 
                            <<" " << c.r
                            <<" " << c.c
                            <<" " << am_num[an]
                            <<" "<<c.block_size
                            <<" "<< (c.dc)
                            <<" "<<(c.am);
  
                            // warmup();
                            // hipEvent_t start, stop;
                            // step access mode 
                            if (_am == 1) {
                                for (int s = 0; s < step_num; ++s) {
                                    c.step = step[s];

                                    if (s != 0)
                                    {
                                        outfile<<endl<<(c.df)
                                        << " Global:"
                                        <<" "<< c.size 
                                        <<" " << c.r
                                        <<" " << c.c
                                        <<" " << am_num[an]
                                        <<" "<<c.block_size
                                        <<" "<< (c.dc)
                                        <<" "<<(c.am);
                                    }

                                    outfile << " " << step[s];

                                    // hipEventCreate(&start);
                                    // hipEventCreate(&stop);
                                    // hipEventRecord(start, 0);
                                    // for (int i = 0; i < LOOP; i++) 
                                        // c.global_run();
                                    // hipEventRecord(stop, 0);
                                    // hipEventSynchronize(stop);
                                    // hipEventElapsedTime(&runTime, start, stop);
                                    // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                                }
                            } else {
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.global_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        }
                }
    
    // 2D constant
    // for (int cs = 0; cs < con_size_num; cs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++)
                for (int cn = 0; cn < col_num; cn++) {
                    Case c;
                    c.df = df_2D;
                    c.size = 512;//con_size[cs];
                    c.c = col[cn];  
                    c.r = (c.size + c.c - 1) / c.c;
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    c.dc = (enum data_content)_dc;
                    c.am = (enum access_mode)_am;
                    // c.initData();
                    for (int an = 0; an < am_num_num; an++)
                        for (int bs= 0; bs < block_size_num; bs++)
                        {
                            c.am_num = am_num[an];
                            c.block_size = block_size[bs];

                            outfile<<endl<<(c.df)
                            << " Constant:"
                            <<" "<< c.size 
                            <<" " << am_num[an]
                            <<" "<<c.block_size
                            <<" "<< (c.dc)
                            <<" "<<(c.am);

                            // warmup();
                            // hipEvent_t start, stop;
                            // step access mode 
                            if (_am == 1) {
                                for (int s = 0; s < step_num; ++s) {
                                    c.step = step[s];

                                    if (s != 0)
                                    {
                                        outfile<<endl<<(c.df)
                                        << " Constant:"
                                        <<" "<< c.size 
                                        <<" " << am_num[an]
                                        <<" "<<c.block_size
                                        <<" "<< (c.dc)
                                        <<" "<<(c.am);
                                    }

                                    outfile << " " << step[s];

                                    // hipEventCreate(&start);
                                    // hipEventCreate(&stop);
                                    // hipEventRecord(start, 0);
                                    // for (int i = 0; i < LOOP; i++) 
                                        // c.constant_run();
                                    // hipEventRecord(stop, 0);
                                    // hipEventSynchronize(stop);
                                    // hipEventElapsedTime(&runTime, start, stop);
                                    // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                                }
                            } else {
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.constant_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        }
                }
  */ 
    // 2D shared
    for (int ss = 0; ss < shared_size_num; ss++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++)
                for (int cn = 0; cn < col_num; cn++) {
                    Case c;
                    c.df = df_2D;
                    c.size = shared_size[ss];
                    c.c = col[cn];  
                    c.r = (c.size + c.c - 1) / c.c;
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    c.dc = (enum data_content)_dc;
                    c.am = (enum access_mode)_am;
                    // c.initData();
                    for (int an = 0; an < am_num_num; an++)
                        for (int bs= 1; bs < block_size_num; bs++) {
                            c.block_size = block_size[bs];
                            c.am_num = am_num[an];

                            outfile<<endl<<(c.df)
                            << " Shared:"
                            <<" "<< c.size 
                            <<" " << c.r
                            <<" " << c.c
                            <<" " << am_num[an]
                            <<" "<<c.block_size
                            <<" "<< (c.dc)
                            <<" "<<(c.am);
                            
                            // warmup();
                            // hipEvent_t start, stop;
                            // step access mode 
                            if (_am == 1) {
                                for (int s = 0; s < step_num; ++s) {
                                    c.step = step[s];

                                    if (s != 0)
                                    {
                                        outfile<<endl<<(c.df)
                                        << " Shared:"
                                        <<" "<< c.size 
                                        <<" " << am_num[an]
                                        <<" "<<c.block_size
                                        <<" "<< (c.dc)
                                        <<" "<<(c.am);
                                    }

                                    outfile << " " << step[s];

                                    // hipEventCreate(&start);
                                    // hipEventCreate(&stop);
                                    // hipEventRecord(start, 0);
                                    // for (int i = 0; i < LOOP; i++) 
                                        // c.shared_run();
                                    // hipEventRecord(stop, 0);
                                    // hipEventSynchronize(stop);
                                    // hipEventElapsedTime(&runTime, start, stop);
                                    // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                                }
                            } else {
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.shared_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        }
                }
   /* 
    // Tree Global
    for (int gs = 0; gs < global_size_num; gs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_tree;
                c.size = global_size[gs];
                c.thread_num = c.size;         // 线程数与数据量大小相同
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();
                for (int an = 0; an < am_num_num; an++) 
                    for (int bs= 0; bs < block_size_num; bs++){
                        c.am_num = am_num[an];
                        c.block_size = block_size[bs];

                        outfile<<endl<<(c.df)
                        << " Global:"
                        <<" "<< c.size 
                        <<" " << am_num[an]
                        <<" "<<c.block_size
                        <<" "<< (c.dc)
                        <<" "<<(c.am);


                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    outfile<<endl<<(c.df)
                                    << " Global:"
                                    <<" "<< c.size 
                                    <<" " << am_num[an]
                                    <<" "<<c.block_size
                                    <<" "<< (c.dc)
                                    <<" "<<(c.am);
                                }

                                outfile << " " << step[s];

                                // warmup();
                                // hipEvent_t start, stop;
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.global_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {

                            // warmup();
                            // hipEvent_t start, stop;
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.global_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }
    // Tree constant
    // for (int cs = 0; cs < con_size_num; cs++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_tree;
                c.size = 512;//  这里每一次都需要手动设置
                c.thread_num = c.size; // 线程数与数据量大小相同
                
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();
                for (int an = 0; an < am_num_num; an++)
                    for (int bs= 0; bs < block_size_num; bs++) {
                        c.am_num = am_num[an];
                        c.block_size = block_size[bs];

                        outfile<<endl<<(c.df)
                        << " Constant:"
                        <<" "<< c.size 
                        <<" " << am_num[an]
                        <<" "<<c.block_size
                        <<" "<< (c.dc)
                        <<" "<<(c.am);

                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    outfile<<endl<<(c.df)
                                    << " Constant:"
                                    <<" "<< c.size 
                                    <<" " << am_num[an]
                                    <<" "<<c.block_size
                                    <<" "<< (c.dc)
                                    <<" "<<(c.am);
                                }

                                outfile << " " << step[s];
                                
                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.constant_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.constant_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }
    
    // Tree shared
    // 在这里要重新设计大小，因为sizeof(tree) = 4+4+sizof(DATA_TYPE)，
    // 同时还要考虑结构体对齐问题
    // DATA_TYPE = unsigned char时, sizeof(tree) = 12B，
    const int tree_shared_size_num = 4;
    const int tree_shared_size[tree_shared_size_num] = {256, 512, 1024, 2048};

    for (int tss = 0; tss < tree_shared_size_num; tss++)
        for (int _dc= 0; _dc < dc; _dc++)
            for (int _am = 0; _am < am; _am++) {
                Case c;
                c.df = df_tree;
                c.size = tree_shared_size[tss];
                c.thread_num = c.size;         // 线程数与数据量大小相同
                c.dc = (enum data_content)_dc;
                c.am = (enum access_mode)_am;
                // c.initData();
                for (int an = 0; an < am_num_num; an++) 
                    for (int bs= 0; bs < block_size_num; bs++){
                        c.am_num = am_num[an];
                        c.block_size = block_size[bs];

                        outfile<<endl<<(c.df)
                        << " Shared:"
                        <<" "<< c.size 
                        <<" " << am_num[an]
                        <<" "<<c.block_size
                        <<" "<< (c.dc)
                        <<" "<<(c.am);

                        // warmup();
                        // hipEvent_t start, stop;
                        // step access mode 
                        if (_am == 1) {
                            for (int s = 0; s < step_num; ++s) {
                                c.step = step[s];

                                if (s != 0)
                                {
                                    outfile<<endl<<(c.df)
                                    << " Shared:"
                                    <<" "<< c.size 
                                    <<" " << am_num[an]
                                    <<" "<<c.block_size
                                    <<" "<< (c.dc)
                                    <<" "<<(c.am);
                                }

                                outfile << " " << step[s];

                                // hipEventCreate(&start);
                                // hipEventCreate(&stop);
                                // hipEventRecord(start, 0);
                                // for (int i = 0; i < LOOP; i++) 
                                    // c.shared_run();
                                // hipEventRecord(stop, 0);
                                // hipEventSynchronize(stop);
                                // hipEventElapsedTime(&runTime, start, stop);
                                // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                            }
                        } else {
                            // hipEventCreate(&start);
                            // hipEventCreate(&stop);
                            // hipEventRecord(start, 0);
                            // for (int i = 0; i < LOOP; i++) 
                                // c.shared_run();
                            // hipEventRecord(stop, 0);
                            // hipEventSynchronize(stop);
                            // hipEventElapsedTime(&runTime, start, stop);
                            // outfile << " runTime=" << (runTime) / LOOP << " ms" << endl;
                        }
                    }
            }
*/

    outfile.close();
}

/*
一维数组，内容为随机分布，访问方式为随机、正态，内存位置为Global和Shared。
shared memory一个线程拷贝一个数据，一个线程访问多个数据，1,2,4,6,8
数据大小：1k，10k，40k

探究：一个线程访问多少数据才能掩盖数据从全局内存拷贝到共享内存的时间消耗
*/ 
void generate_params1()
{
    char* filename = "params.txt";
    remove(filename);
    ofstream outfile("params.txt");

    // global 测试数据大小
    const int global_size_num = 4; // 数组大小;
    const int global_size[global_size_num] = {1024, 4096, 10240, 40960};

    // constant 和 
    // const int con_size_num = 4;
    // const int con_size[con_size_num] = {512, 1024, 4096, 10240};
    // shared 测试数据大小,这里的大小设置受 DATA_TYPE 的影响。
    const int shared_size_num = 4;
    const int shared_size[shared_size_num] = {1024, 4096, 10240, 40960};
    // block size
    const int block_size_num = 4;
    const int block_size[block_size_num] = {128, 256, 512, 1024};
    // 六种数据内部分布，具体见common.h
    const int dc = 6;
    // 七种访问数据类型，具体见common.h
    const int am = 7;
    // 每个线程访问多少数据
    const int am_num_num = 8;
    const int am_num[am_num_num] = {1, 2, 4, 6, 8, 12, 16, 24};

    // 1D Global
    for (int gs = 0; gs < global_size_num; gs++)
        // for (int _dc= 0; _dc < dc; _dc++)
        for (int _am = 2; _am < 4; _am++) {
            Case c;
            c.df = df_1D;
            c.size = global_size[gs];
            // 内部分布为随机
            c.dc = dc_random;
            c.am = (enum access_mode)_am;
            // c.initData();

            for (int bs= 0; bs < block_size_num; bs++)
                for (int an = 0; an < am_num_num; an++) {   
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    c.block_size = block_size[bs];   
                    c.am_num = am_num[an];

                    outfile<<endl<<(c.df)
                    << " Global:"
                    <<" "<< c.size 
                    <<" " << am_num[an]
                    <<" "<<c.block_size
                    <<" "<< (c.dc)
                    <<" "<<(c.am);
                }
        }

    // 1D shared
    for (int ss = 0; ss < shared_size_num; ss++)
        // for (int _dc= 0; _dc < dc; _dc++)
        for (int _am = 2; _am < 4; _am++) {
            Case c;
            c.df = df_1D;
            c.size = shared_size[ss];
            c.dc = dc_random;
            c.am = (enum access_mode)_am;
            // c.initData();

            for (int bs= 0; bs < block_size_num; bs++)
                for (int an = 0; an < am_num_num; an++) {   
                    c.thread_num = c.size;         // 线程数与数据量大小相同
                    c.block_size = block_size[bs];   
                    c.am_num = am_num[an];

                    outfile<<endl<<(c.df)
                    << " Shared:"
                    <<" "<< c.size 
                    <<" " << am_num[an]
                    <<" "<<c.block_size
                    <<" "<< (c.dc)
                    <<" "<<(c.am);
                }
        }

    outfile.close();
}
