#include <iostream>
/*#include "Case.h"*/
#include "GenerateParams.h"
#include <stdlib.h>
#include <string>
using namespace std;

int main(int argc, char const *argv[])
{
    generate_params1();
    /*return 0;*/

    Case c;
    unsigned int index = 1;
    c.df = (enum data_form)atoi(argv[index ++]);
    index ++;
    if (string(argv[2]) == "Constant:")
        c.size = CONSTANT_SIZE;
    else 
        c.size = atoi(argv[index ++]);
    if (c.df == df_2D) {
        if (string(argv[2]) != "Constant:") {
            c.c = atoi(argv[index ++]);  
            c.r = atoi(argv[index ++]); 
        } else {
            c.c = CONSTANT_SIZE / CONSTANT_2D_ROW;  
            c.r = CONSTANT_2D_ROW; 
        }
    }
    c.am_num = atoi(argv[index ++]);
    c.block_size = atoi(argv[index ++]);
    c.dc = (enum data_content)atoi(argv[index ++]);
    c.am = (enum access_mode)atoi(argv[index ++]);
    c.thread_num = c.size;         // 线程数与数据量大小相同
      
    if (c.am == am_step)
        c.step = atoi(argv[index ++]);

    cout<<endl<<EnumToString(c.df) << " " 
        << argv[2] 
        <<" size="<< c.size;
        if (c.df == df_2D)
            cout<<" r=" << c.r
            <<",c=" << c.c;
        cout<<" access_num_per_thread=" << c.am_num
        <<" block_size="<<c.block_size
        <<" data_content="<< EnumToString(c.dc)
        <<" access_mode="<<EnumToString(c.am);
        if (c.am == am_step)
            cout << " step=" << c.step;
    
    warmup();
    c.initData();
    float runTime = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    if (string(argv[2]) == "Global:") {
         c.global_run();
        /*cout << "c.global_run();" << endl;*/
    } else if (string(argv[2]) == "Shared:") {
         c.shared_run();
        /*cout << "c.shared_run();" << endl;*/
    } else if (string(argv[2]) == "Constant:"){
         c.constant_run();
        /*cout << "c.constant_run();" << endl;*/
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);

    
    cout << " runTime=" << (runTime)  << " ms" << endl;
    

    return 0;
}
