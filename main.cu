#include <iostream>
/*#include "Case.h"*/
#include "GenerateParams.h"
#include <stdlib.h>
#include <string>
using namespace std;

int main(int argc, char const *argv[])
{
    // for (int i = 0; i < argc; ++i)
    // {
    //     cout << argv[i] << " ";
    // }
    // cout << endl;
    // return 0;
    /*generate_params();*/
    /*return 0;*/

    Case c;
    unsigned int index = 1;
    c.df = (enum data_form)atoi(argv[index ++]);
    index ++;
    c.size = atoi(argv[index ++]);
    if (c.df == df_2D) {
        c.c = atoi(argv[index ++]);  
        c.r = atoi(argv[index ++]); 
    }
    c.am_num = atoi(argv[index ++]);
    c.block_size = atoi(argv[index ++]);
    c.dc = (enum data_content)atoi(argv[index ++]);
    c.am = (enum access_mode)atoi(argv[index ++]);
    c.thread_num = c.size;         // 线程数与数据量大小相同
      
    if (c.am == am_step)
        c.step = atoi(argv[index ++]);
    
    /*c.print();*/

    warmup();
    c.initData();
    float runTime = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    if (string(argv[2]) == "Global:") {
         c.global_run();
        /*cout << "c.global_run();" << endl;*/
    } else if (string(argv[2]) == "Shared:") {
         c.shared_run();
        /*cout << "c.shared_run();" << endl;*/
    } else if (string(argv[2]) == "Constant:"){
         c.constant_run();
        /*cout << "c.constant_run();" << endl;*/
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);

    cout<<endl<<EnumToString(c.df) << " " 
        << argv[2] 
        <<" size="<< c.size;
        if (c.df == df_2D)
            cout<<" r=" << c.r
            <<",c=" << c.c;
        cout<<" access_num_per_thread=" << c.am_num
        <<" block_size="<<c.block_size
        <<" data_content="<< EnumToString(c.dc)
        <<" access_mode="<<EnumToString(c.am);
        if (c.am == am_step)
            cout << " step=" << c.step;
    cout << " runTime=" << (runTime)  << " ms" << endl;
    

    return 0;
}
