#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
using namespace std;

/* 树节点 */ 
class Node
{
public:
    Node* left;
    Node* right;
    DATA_TYPE data;
};

__global__ void testKer()
{
    printf("Node's size in GPU: %d", sizeof(Node));
}

int main()
{
    cout << "Node's size in CPU:" << sizeof(Node) << endl;

    testKer<<<1, 1>>>();

    hipDeviceReset();
    return 0;
}