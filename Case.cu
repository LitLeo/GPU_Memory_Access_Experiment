#include "hip/hip_runtime.h"
#include "Case.h"

// 设置二维默认线程数
#define DEF_BLOCK_X  32
#define DEF_BLOCK_Y   8

// 全局内容的大小必须提前设置，对于不同数据大小的测试只能一次次手动改了
__constant__ DATA_TYPE constant_data1D[1];
/*__constant__ DATA_TYPE constant_data1D[CONSTANT_SIZE];*/
__constant__ DATA_TYPE constant_data2D[1][1];
/*__constant__ DATA_TYPE constant_data2D[CONSTANT_SIZE/CONSTANT_2D_ROW][CONSTANT_2D_ROW];*/
// 每个 node 的大小为 12 B，所以在设置大小时一定要注意不能超过 constant memory 的大小
/*__constant__ Node constant_treeNodes[1];*/
__constant__ Node constant_treeNodes[CONSTANT_SIZE];

// 根据数据组织形式、数据大小和数据内容形式初始化数据
int Case::initData()
{
    // 数据组织形式
    switch (this->df)
    {
    case df_1D:
        this->data1D = new DATA_TYPE[this->size];
        // 数据内容形式初始化数据
        switch (this->dc)
        {
        case dc_exponential:
            // 设置 lambda = 10
            if (exponential(10 ,MIN, MAX, this->size, this->data1D))
                return -1;
            break;
        case dc_geometric:
            // 设置 p = 0.5
            if (geometric(0.5 ,MIN, MAX, this->size, this->data1D))
                return -1;
            break;
        case dc_poisson:
            // 设置 Lambda = 05
            if (poisson(MIN, MAX, 5, this->size, this->data1D))
                return -1;
            break;
        case dc_random:
            if (random(MIN, MAX, this->size, this->data1D))
                return -1;
            break;
        case dc_standard_normal:
            if (standard_normal(MIN, MAX, MAX - MIN, 1,  this->size, this->data1D))
                return -1;
            break;
        case dc_uniform:
            if (uniform(MIN, MAX, this->size, this->data1D))
                return -1;
            break;
        }
        break;
    case df_2D:
    {
        if (this->r * this->c != this->size)
            this->size = this->r * this->c;
        this->data2D.data = new DATA_TYPE[this->size];
        this->data2D.rows = this->r;
        this->data2D.cols = this->c;
        // 数据内容形式初始化数据
        switch (this->dc)
        {
        case dc_exponential:
            if (exponential(10 ,MIN, MAX, this->size, this->data2D.data))
                return -1;
            break;
        case dc_geometric:
            // 设置 p = 0.5
            if (geometric(0.5 ,MIN, MAX, this->size, this->data2D.data))
                return -1;
            break;
        case dc_poisson:
            // 设置 Lambda = 05
            if (poisson(MIN, MAX, 5, this->size, this->data2D.data))
                return -1;
            break;
        case dc_random:
            if (random(MIN, MAX, this->size, this->data2D.data))
                return -1;
            break;
        case dc_standard_normal:
            if (standard_normal(MIN, MAX, MAX - MIN, 1,  this->size, this->data2D.data))
                return -1;
            break;
        case dc_uniform:
            if (uniform(MIN, MAX, this->size, this->data2D.data))
                return -1;
            break;
        }

        break;
    }
    case df_tree:
        this->tree= new Tree(size);

        switch (this->dc)
        {
        case dc_exponential:
            if (exponential(10 ,MIN, MAX, this->size, this->tree->nodes))
                return -1;
            break;
        case dc_geometric:
            // 设置 p = 0.5
            if (geometric(0.5 ,MIN, MAX, this->size, this->tree->nodes))
                return -1;
            break;
        case dc_poisson:
            // 设置 Lambda = 05
            if (poisson(MIN, MAX, 5, this->size, this->tree->nodes))
                return -1;
            break;
        case dc_random:
            if (random(MIN, MAX, this->size, this->tree->nodes))
                return -1;
            break;
        case dc_standard_normal:
            if (standard_normal(MIN, MAX, MAX - MIN, 1,  this->size, this->tree->nodes))
                return -1;
            break;
        case dc_uniform:
            if (uniform(MIN, MAX, this->size, this->tree->nodes))
                return -1;
            break;
        }

        break;
    }
    // 当访问方式为分布访问时（即不是顺序访问和step访问）
    // 需要生成访问下标数据
    if (this->am != am_sequential && this->am != am_step) {
        this->host_am_data = new int[this->size];
        switch(this->am)
        {
            case am_random:
            if (random(0, this->size, host_am_data))
                return -1;
            break;
            case am_standard_normal:
            if (standard_normal(0, this->size, this->size, 1,  host_am_data))
                return -1;
            break;
            case am_poisson:
            if (poisson(0, this->size, 5, host_am_data))
                return -1;
            break;

            case am_geometric:
            if (geometric(0.5, 0, this->size, host_am_data))
                return -1;
           
            break;
            case am_exponential:
            if (exponential(10 ,0, this->size, host_am_data))
                return -1;
            break;
        }
    }

    return 0;
}

// 全局内存
// data1D,dev_out,am_data的size与线程数相同
/*顺序访问数据*/
static __global__ void _d1DGloalSequentialKer(DATA_TYPE *data1D, DATA_TYPE* dev_out, int am_num, int size)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i)
        dev_out[index] += data1D[(index + i) % size]; 

}
/*step方式访问数据*/
static __global__ void _d1DGloalStepKer(DATA_TYPE *data1D, DATA_TYPE* dev_out, int step, int am_num, int size)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i)
        dev_out[index] += data1D[(index + i * step) % size]; 
}
/*根据不同的分布进行访问，标准正态分布、泊松分布、指数分布、几何分布
因为生成 am_data 需要使用随机数，所以 am_data 需要在核函数外生成, am_data内数据值域为[0, size)*/ 
static __global__ void _d1DGloalCommonKer(DATA_TYPE *data1D, DATA_TYPE* dev_out, int am_num, 
                                          int* am_data, int size)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i)
        dev_out[i] += data1D[am_data[(index + i) % size]];
}

// 共享内存
// 将数据全部拷贝到共享内存中
// 数据量与线程的关系
static __global__ void _d1DSharedSequentialKer(DATA_TYPE *data1D, DATA_TYPE* dev_out, int am_num, 
                                               int size, int copy_num_per_thread)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= size)
        return ;

    // 将数据拷贝到共享内存中, shared memory size = data size，
    // 大小需要在核函数外设置
    extern __shared__ DATA_TYPE sharedData[];
    // 一个线程拷贝 (data_size + T - 1) / T 个数据
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copt_index = index * copy_num_per_thread + i;
        if (copt_index < size)
            sharedData[copt_index] = data1D[copt_index];
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i)
        dev_out[index] += sharedData[(index + i) % size]; 
}
static __global__ void _d1DSharedStepKer(DATA_TYPE *data1D, DATA_TYPE* dev_out, int step, int am_num,
                                         int size, int copy_num_per_thread)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= size)
        return ;

    // 将数据拷贝到共享内存中
    extern __shared__ DATA_TYPE sharedData[];
    // 一个线程拷贝 (data_size + T - 1) / T 个数据
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copt_index = index * copy_num_per_thread + i;
        if (copt_index < size)
            sharedData[copt_index] = data1D[copt_index];
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i)
        dev_out[index] += sharedData[(index + i * step) % size]; 
}
static __global__ void _d1DSharedCommonKer(DATA_TYPE *data1D, DATA_TYPE* dev_out, int am_num, int size, int* am_data,
                                           int copy_num_per_thread)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= size)
        return ;

    // 将数据拷贝到共享内存中
    extern __shared__ DATA_TYPE sharedData[];
    // 一个线程拷贝 (data_size + T - 1) / T 个数据
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copt_index = index * copy_num_per_thread + i;
        if (copt_index < size)
            sharedData[copt_index] = data1D[copt_index];
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i)
        dev_out[index] += sharedData[am_data[(index + i)%size] % size];
}

// 常量内存
// constant_data1D
static __global__ void _d1DConstantSequentialKer(DATA_TYPE* dev_out, int am_num, int size)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i)
        dev_out[index] += constant_data1D[(index + i) % size]; 
}
static __global__ void _d1DConstantStepKer(DATA_TYPE* dev_out, int step, int am_num, int size)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i)
        dev_out[index] += constant_data1D[(index + i * step) % size]; 
}
static __global__ void _d1DConstantCommonKer(DATA_TYPE* dev_out, int am_num, int* am_data, int size)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i)
        dev_out[index] += constant_data1D[am_data[(index + i)%size]];
}

// ！！！！需要考虑越界问题
// 一维与二维的区别，在根据下标访问时，需要考虑 pitchByte 问题
// 访问二维数组的方式：输入是相对应的一维下标 index，将 index 由二维矩阵的 width 转变为 row, col，
// 根据偏移量 offset 更新 col, row（此时要考虑col + offset >= width 问题），从而得到数据的索引为
// index = row * pitchBytes + col.
// 输出数据的大小 == thread num == data2D.cols * data2D.rows
static __global__ void _d2DGloalSequentialKer(Data2D data2D, DATA_TYPE* dev_out, int am_num)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= data2D.cols || r >= data2D.rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * data2D.cols + c;
    for (int i = 0; i < am_num; ++i) {
        int temp_c = c + i;
        // 进行越界判断，并重新计算下标
        if (temp_c >= data2D.cols) {
            r += (temp_c) / data2D.cols; 
            c = (temp_c) % data2D.cols;
            if (r >= data2D.rows)
                r = data2D.rows - 1;
        }
        dev_out[index] += data2D.data[r * data2D.pitchBytes + c]; 
    }
}
/*step 访问时，col * step*/
static __global__ void _d2DGloalStepKer(Data2D data2D, DATA_TYPE* dev_out, int step, int am_num)
{
    // 获得线程索引
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= data2D.cols || r >= data2D.rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * data2D.cols + c;
    for (int i = 0; i < am_num; ++i){
        int temp_c = c + i * step;
        // 进行越界判断，并重新计算下标
        if (temp_c >= data2D.cols) {
            r += (temp_c) / data2D.cols; 
            c = (temp_c) % data2D.cols;
            if (r >= data2D.rows)
                r = data2D.rows - 1;
        }
        dev_out[index] += data2D.data[(r * data2D.pitchBytes + c)];
    }
}
static __global__ void _d2DGloalCommonKer(Data2D data2D, DATA_TYPE* dev_out, int am_num, int* am_data)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= data2D.cols || r >= data2D.rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * data2D.cols + c;
    // 数据大小
    unsigned int size = data2D.cols * data2D.rows;
    for (int i = 0; i < am_num; ++i){
        // 从 am_data 中根据线程下标得到访问数据下标
        // 并重新计算 r c
        unsigned int temp_am_data = am_data[(index + i) % size];
        c = temp_am_data % data2D.cols;
        r = temp_am_data / data2D.rows;
        if (r >= data2D.rows)
            r = data2D.rows - 1;
        dev_out[index] += data2D.data[r * data2D.pitchBytes + c];
    }
}

static __global__ void _d2DSharedSequentialKer(Data2D data2D, DATA_TYPE* dev_out, int am_num, int copy_num_per_thread)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= data2D.cols || r >= data2D.rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * data2D.cols + c;
    // 数据大小
    unsigned int size = data2D.cols * data2D.rows;
    // 申请 shared memory，并拷贝数据
    extern __shared__ DATA_TYPE sharedData[];
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copt_index = index * copy_num_per_thread + i;
        if (copt_index < size) {
            // 又要考虑溢出问题……
            r = r + (c+i) / data2D.cols;
            if (r >= data2D.rows)
                r = data2D.rows - 1;
            sharedData[copt_index] = data2D.data[r * data2D.pitchBytes + (c+i) % data2D.cols];
        }
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i) {
        // 因为 shared 是一维数组，所以在这里并不需要考虑溢出问题
        dev_out[index] += sharedData[(r * data2D.cols + c + i) % size]; 
    }
}
static __global__ void _d2DSharedStepKer(Data2D data2D, DATA_TYPE* dev_out, int step, int am_num, int copy_num_per_thread)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= data2D.cols || r >= data2D.rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * data2D.cols + c;
    // 数据大小
    unsigned int size = data2D.cols * data2D.rows;
    // 申请 shared memory，并拷贝数据
    extern __shared__ DATA_TYPE sharedData[];
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copt_index = index * copy_num_per_thread + i;
        if (copt_index < size) {
            // 又要考虑溢出问题……
            r = r + (c+i) / data2D.cols;
            if (r >= data2D.rows)
                r = data2D.rows - 1;
            sharedData[copt_index] = data2D.data[r  * data2D.pitchBytes + (c+i) % data2D.cols];
        }
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i) {
        // 因为 shared 是一维数组，所以在这里并不需要考虑溢出问题
        dev_out[index] += sharedData[(r * data2D.cols + c + i * step) % size]; 
    }
}
static __global__ void _d2DSharedCommonKer(Data2D data2D, DATA_TYPE* dev_out, int am_num, int* am_data, int copy_num_per_thread)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= data2D.cols || r >= data2D.rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * data2D.cols + c;
    // 数据大小
    unsigned int size = data2D.cols * data2D.rows;
    // 申请 shared memory，并拷贝数据
    extern __shared__ DATA_TYPE sharedData[];
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copt_index = index * copy_num_per_thread + i;
        if (copt_index < size) {
            // 又要考虑溢出问题……
            r = r + (c+i) / data2D.cols;
            if (r >= data2D.rows)
                r = data2D.rows - 1;
            sharedData[copt_index] = data2D.data[r * data2D.pitchBytes + (c+i) % data2D.cols];
        }
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i) {
        // 从 am_data 中根据线程下标得到访问数据下标
        // 并重新计算 r c
        unsigned int temp_am_data = am_data[(index + i) % size];
        c = temp_am_data % data2D.cols;
        r = temp_am_data / data2D.rows;
        if (r >= data2D.rows)
            r = data2D.rows - 1;
        dev_out[index] += sharedData[r * data2D.cols + c];
    }
}
/*
申请一个二维的常量内存
常量内存需要设置成二维的，设成成一维的话，就跟一维数组没有区别了……
 */
static __global__ void _d2DConstantSequentialKer(DATA_TYPE* dev_out, int am_num, int cols, int rows)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * cols + c;
    // 数据大小
    // unsigned int size = cols * rows;

    for (int i = 0; i < am_num; ++i) {
        r += (c+i) / cols;
        c = (c+i) % cols;
        if (r >= rows)
            r = rows - 1;
        dev_out[index] += constant_data2D[r][c]; 
    }

}
static __global__ void _d2DConstantStepKer(DATA_TYPE* dev_out, int step, int am_num, int cols, int rows)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * cols + c;
    // 数据大小
    // unsigned int size = cols * rows;

    for (int i = 0; i < am_num; ++i) {
        r += (c + i*step) / cols;
        c = (c + i*step) % cols;
        if (r >= rows)
            r = rows - 1;
        dev_out[index] += constant_data2D[r][c]; 
    }
}
static __global__ void _d2DConstantCommonKer(DATA_TYPE* dev_out, int am_num, int* am_data, int cols, int rows)
{
    // 获得线程索引
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return ;
    // 计算输出数据的下标
    unsigned int index = r * cols + c;
    // 数据大小
    // unsigned int size = cols * rows;

    for (int i = 0; i < am_num; ++i) {
        r += am_data[index] / cols;
        c = am_data[index] % cols;
        if (r >= rows)
            r = rows - 1;
        dev_out[index] += constant_data2D[r][c]; 
    }
}

/*
树的访问方式：
根据下标访问某一个节点，然后求该节点的子节点之和
树结构本质上是一个一维数组
*/
static __global__ void _treeGloalSequentialKer(Tree tree, int am_num, DATA_TYPE* dev_out)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= tree.num)
        return ;
    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += tree.nodes[tempindex % tree.num].data;
        tempindex ++;
        dev_out[index] += tree.nodes[tempindex % tree.num].data;     
    }
}
static __global__ void _treeGloalStepKer(Tree tree, int step, int am_num, DATA_TYPE* dev_out)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= tree.num)
        return ;
    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += tree.nodes[tempindex % tree.num].data;
        tempindex ++;
        dev_out[index] += tree.nodes[tempindex % tree.num].data;     
    }
}
static __global__ void _treeGloalCommonKer(Tree tree, int am_num, DATA_TYPE* dev_out, int* am_data)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= tree.num)
        return ;
    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += tree.nodes[tempindex % tree.num].data;
        tempindex ++;
        dev_out[index] += tree.nodes[tempindex % tree.num].data;     
    }
}

static __global__ void _treeSharedSequentialKer(Tree tree, int am_num, DATA_TYPE* dev_out, int copy_num_per_thread)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= tree.num)
        return ;
    // 将数据拷贝到共享内存中,
    // 大小需要在核函数外设置
    extern __shared__ Node sharedNodes[];
    // 一个线程拷贝 (data_size + T - 1) / T 个数据
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copy_index = index * copy_num_per_thread + i;
        if (copy_index < tree.num) {
            sharedNodes[copy_index].left = tree.nodes[copy_index].left;
            sharedNodes[copy_index].right = tree.nodes[copy_index].right;
            sharedNodes[copy_index].data = tree.nodes[copy_index].data;
        }
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += sharedNodes[tempindex % tree.num].data;
        tempindex ++;
        dev_out[index] += sharedNodes[tempindex % tree.num].data;     
    }
}
static __global__ void _treeSharedStepKer(Tree tree, int step, int am_num, DATA_TYPE* dev_out, int copy_num_per_thread)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= tree.num)
        return ;
    // 将数据拷贝到共享内存中,
    // 大小需要在核函数外设置
    extern __shared__ Node sharedNodes[];
    // 一个线程拷贝 (data_size + T - 1) / T 个数据
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copy_index = index * copy_num_per_thread + i;
        if (copy_index < tree.num) {
            sharedNodes[copy_index].left = tree.nodes[copy_index].left;
            sharedNodes[copy_index].right = tree.nodes[copy_index].right;
            sharedNodes[copy_index].data = tree.nodes[copy_index].data;
        }
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += sharedNodes[tempindex % tree.num].data;
        tempindex ++;
        dev_out[index] += sharedNodes[tempindex % tree.num].data;     
    }
}
static __global__ void _treeSharedCommonKer(Tree tree, int am_num, DATA_TYPE* dev_out, int* am_data, int copy_num_per_thread)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= tree.num)
        return ;
    // 将数据拷贝到共享内存中,
    // 大小需要在核函数外设置
    extern __shared__ Node sharedNodes[];
    // 一个线程拷贝 (data_size + T - 1) / T 个数据
    for (int i = 0; i < copy_num_per_thread; ++i) {
        // 计算要拷贝数据的下标
        int copy_index = index * copy_num_per_thread + i;
        if (copy_index < tree.num) {
            sharedNodes[copy_index].left = tree.nodes[copy_index].left;
            sharedNodes[copy_index].right = tree.nodes[copy_index].right;
            sharedNodes[copy_index].data = tree.nodes[copy_index].data;
        }
    }
    __syncthreads();

    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += sharedNodes[tempindex % tree.num].data;
        tempindex ++;
        dev_out[index] += sharedNodes[tempindex % tree.num].data;     
    }
}

static __global__ void _treeConstantSequentialKer(int size, int am_num, DATA_TYPE* dev_out)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;

    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += constant_treeNodes[tempindex % size].data;
        tempindex ++;
        dev_out[index] += constant_treeNodes[tempindex % size].data;     
    }
}
static __global__ void _treeConstantStepKer(int size, int step, int am_num, DATA_TYPE* dev_out)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += constant_treeNodes[tempindex % size].data;
        tempindex ++;
        dev_out[index] += constant_treeNodes[tempindex % size].data;     
    }
}
static __global__ void _treeConstantCommonKer(int size, int am_num, DATA_TYPE* dev_out, int* am_data)
{
    // 获得线程索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return ;
    for (int i = 0; i < am_num; ++i) {
        int tempindex = (index + i) * 2 + 1;
        dev_out[index] += constant_treeNodes[tempindex % size].data;
        tempindex ++;
        dev_out[index] += constant_treeNodes[tempindex % size].data;     
    }
}

// 申请device内存
// 内存拷贝
// 核函数
int Case::global_run()
{
    // 错误代码
    hipError_t cuerrcode;

    // 申请device内存
    DATA_TYPE *d_data1D = NULL;
    Data2D d_data2D;
    d_data2D.data = NULL;
    Tree d_tree;  // 无参构造函数，没有申请节点空间
    d_tree.nodes = NULL;

    DATA_TYPE *dev_out1D = NULL;
    DATA_TYPE *dev_out2D = NULL;
    DATA_TYPE *dev_outTree = NULL;

    int *dev_am_data = NULL;

    // 根据内部数据分布，申请空间并拷贝数据
    switch (this->df)
    {
    case df_1D:
        // 申请device端空间
        cuerrcode = hipMalloc((void **)&d_data1D, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) 
            return -1;
        // 数据拷贝
        cuerrcode = hipMemcpy(d_data1D, this->data1D, sizeof(DATA_TYPE) * this->size, hipMemcpyHostToDevice);
        if (cuerrcode != hipSuccess) {
            // 数据拷贝出错，返回错误代码前释放申请的空间
            hipFree(d_data1D);
            return -1;
        }
        break;
    case df_2D:
        // 错误情况判断
        if (this->c == -1 || this->r == -1)
            return -2;
        d_data2D.cols = this->c;
        d_data2D.rows = this->r;
        this->size = this->r * this->c;
        cuerrcode = hipMallocPitch((void **)&d_data2D.data, &d_data2D.pitchBytes,
                                      d_data2D.cols * sizeof(DATA_TYPE), d_data2D.rows);
        if (cuerrcode != hipSuccess) 
            return -1;
        cuerrcode = hipMemcpy2D(d_data2D.data, d_data2D.pitchBytes, 
                                   this->data2D.data, d_data2D.pitchBytes,
                                   d_data2D.cols * sizeof(DATA_TYPE), d_data2D.rows,
                                   hipMemcpyHostToDevice);
        if (cuerrcode != hipSuccess) {
            // 数据拷贝出错，返回错误代码前释放申请的空间
            hipFree(d_data2D.data);
            return -1;
        }
        break;
    case df_tree:
        d_tree.num = this->size;
        cuerrcode = hipMalloc((void**)&d_tree.nodes, sizeof(Node) * d_tree.num);
        if (cuerrcode != hipSuccess)
            return -1;
        cuerrcode = hipMemcpy(d_tree.nodes, this->tree->nodes, sizeof(Node) * d_tree.num, hipMemcpyHostToDevice);
        if (cuerrcode != hipSuccess) {
            hipFree(d_tree.nodes);
            return -1;
        }
        break;
    }
    // 根据数据形式和访问方式的不同执行不同的核函数
    switch(this->df) {
    case df_1D:
    {
        // 申请 out 数组空间
        cuerrcode = hipMalloc((void**)&dev_out1D, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) {
            hipFree(d_data1D);
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_out1D, 0, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) {
            hipFree(dev_out1D);
            hipFree(d_data1D);
            return -1;
        }
        // 核函数
        // 计算线程数
        int gridsize_1d, blocksize_1d;
        blocksize_1d = this->block_size;
        gridsize_1d = (this->thread_num + blocksize_1d - 1) / blocksize_1d;
        if (this->am == am_step) {
            _d1DGloalStepKer<<<gridsize_1d, blocksize_1d>>>(d_data1D, dev_out1D, this->step, this->am_num, this->size);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                return -3;
            }
        } else if (this->am == am_sequential) {
            _d1DGloalSequentialKer<<<gridsize_1d, blocksize_1d>>>(d_data1D, dev_out1D, this->am_num, this->size);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                return -3;
            }
        } else {
            // device 端访问下标数据
            int *dev_am_data;
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * this->size, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                hipFree(dev_am_data);
                return -3;
            }
            // hiprandDestroyGenerator(gen);

            _d1DGloalCommonKer<<<gridsize_1d, blocksize_1d>>>(d_data1D, dev_out1D, this->am_num, dev_am_data, this->size);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                hipFree(dev_am_data);
                return -3;
            }
        }
        break;
    }
    case df_2D:
    {
        // 计算二维数组的大小，这里重新计算的是为了方式只测试二维数组时，成员变量size
        // 没有及时更新。
        unsigned int size_2D = d_data2D.cols * d_data2D.rows;
        // 申请 out 数组空间
        cuerrcode = hipMalloc((void**)&dev_out2D, sizeof(DATA_TYPE) * size_2D);
        if (cuerrcode != hipSuccess) {
            hipFree(d_data2D.data);
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_out2D, 0, sizeof(DATA_TYPE) * size_2D);
        if (cuerrcode != hipSuccess) {
            hipFree(dev_out2D);
            hipFree(d_data2D.data);
            return -1;
        }
        // 计算线程数，设置二维网格
        // 默认 block.x为 32
        dim3 gridsize, blocksize;
        blocksize.x = DEF_BLOCK_X; // = 32
        blocksize.y = this->block_size / blocksize.x;
        gridsize.x = (d_data2D.cols + blocksize.x - 1) / blocksize.x;
        gridsize.y = (d_data2D.rows + blocksize.y - 1) / blocksize.y;
        if (this->am == am_sequential) {
            _d2DGloalSequentialKer<<<gridsize, blocksize>>>(d_data2D, dev_out2D, this->am_num);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_out2D);
                hipFree(d_data2D.data);
                return -3;
            }
        } else if (this->am == am_step) {
            _d2DGloalStepKer<<<gridsize, blocksize>>>(d_data2D, dev_out2D, this->step, this->am_num);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_out2D);
                hipFree(d_data2D.data);
                return -3;
            }
        } else {

            // 根据不同访问方式在核函数外生成访问下标
            // P.S. 虽然可以用hiprand在核函数外生成一些分布，但am_geometric分布和am_exponential分布无法生成，
            // 为了考虑性能测试统一，不使用hiprand。
            // 先在 host 端生成下标分布数据，然后再拷贝到 device 端。
            // device 端访问下标数据
            int *dev_am_data;
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(d_data2D.data);
                hipFree(dev_out2D);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * size_2D, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(dev_out2D);
                hipFree(d_data2D.data);
                return -3;
            }

            _d2DGloalCommonKer<<<gridsize, blocksize>>>(d_data2D, dev_out2D, this->am_num, dev_am_data);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(dev_out2D);
                hipFree(d_data2D.data);
                return -3;
            }
        }
        break;
    }
    case df_tree: {
        cuerrcode = hipMalloc((void**)&dev_outTree, sizeof(DATA_TYPE) * this->tree->num);
        if (cuerrcode != hipSuccess) {
            hipFree(d_tree.nodes);
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_outTree, 0, sizeof(DATA_TYPE) * this->tree->num);
        if (cuerrcode != hipSuccess) {
            hipFree(d_tree.nodes);
            hipFree(dev_outTree);
            return -1;
        }
        int gridsize_1d, blocksize_1d;
        blocksize_1d = this->block_size;
        gridsize_1d = (this->thread_num + blocksize_1d - 1) / blocksize_1d;
        // 核函数
        if (this->am == am_sequential) {
            _treeGloalSequentialKer<<<gridsize_1d, blocksize_1d>>>(d_tree, this->am_num, dev_outTree);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                return -3;
            }
        } else if (this->am == am_step) {
            _treeGloalStepKer<<<gridsize_1d, blocksize_1d>>>(d_tree, this->step, this->am_num, dev_outTree);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                return -3;
            }
        } else {
            // device 端访问下标数据
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * this->size, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                hipFree(dev_am_data);
                return -3;
            }

            _treeGloalCommonKer<<<gridsize_1d, blocksize_1d>>>(d_tree, this->am_num, dev_outTree, dev_am_data);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                hipFree(dev_am_data);
                return -3;
            }
        }
        break;
    }
    }

    if (d_data1D == NULL)
        hipFree(d_data1D);
    if (d_data2D.data == NULL)
        hipFree(d_data2D.data);
    if (d_tree.nodes == NULL)
        hipFree(d_tree.nodes);

    if (dev_out1D == NULL)
        hipFree(dev_out1D);
    if (dev_out2D == NULL)
        hipFree(dev_out2D);
    if (dev_outTree == NULL)
        hipFree(dev_outTree);
    if (dev_am_data == NULL)
        hipFree(dev_am_data);

    return 0;
}

int Case::shared_run()
{
    // 错误代码
    hipError_t cuerrcode;

    // 申请device内存
    DATA_TYPE *d_data1D = NULL;
    Data2D d_data2D;
    d_data2D.data = NULL;
    Tree d_tree;  // 无参构造函数，没有申请节点空间
    d_tree.nodes = NULL;
    d_tree.num = this->size;


    DATA_TYPE *dev_out1D = NULL;
    DATA_TYPE *dev_out2D = NULL;
    DATA_TYPE *dev_outTree = NULL;

    int *dev_am_data = NULL;

    // 根据内部数据分布，申请空间并拷贝数据
    switch (this->df)
    {
    case df_1D:
        // 申请device端空间
        cuerrcode = hipMalloc((void **)&d_data1D, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) 
            return -1;
        // 数据拷贝
        cuerrcode = hipMemcpy(d_data1D, this->data1D, sizeof(DATA_TYPE) * this->size, hipMemcpyHostToDevice);
        if (cuerrcode != hipSuccess) {
            // 数据拷贝出错，返回错误代码前释放申请的空间
            hipFree(d_data1D);
            return -1;
        }
        break;
    case df_2D:
        // 错误情况判断
        if (this->c == -1 || this->r == -1)
            return -2;
        d_data2D.cols = this->c;
        d_data2D.rows = this->r;
        cuerrcode = hipMallocPitch((void **)&d_data2D.data, &d_data2D.pitchBytes,
                                      d_data2D.cols * sizeof(DATA_TYPE), d_data2D.rows);
        if (cuerrcode != hipSuccess) 
            return -1;
        cuerrcode = hipMemcpy2D(d_data2D.data, d_data2D.pitchBytes, 
                                   this->data2D.data, d_data2D.pitchBytes,
                                   d_data2D.cols * sizeof(DATA_TYPE), d_data2D.rows,
                                   hipMemcpyHostToDevice);
        if (cuerrcode != hipSuccess) {
            // 数据拷贝出错，返回错误代码前释放申请的空间
            hipFree(d_data2D.data);
            return -1;
        }
        break;
    case df_tree:
        cuerrcode = hipMalloc((void**)&d_tree.nodes, sizeof(Node) * d_tree.num);
        if (cuerrcode != hipSuccess)
            return -1;
        cuerrcode = hipMemcpy(d_tree.nodes, this->tree->nodes, sizeof(Node) * d_tree.num, hipMemcpyHostToDevice);
        if (cuerrcode != hipSuccess)
        {
            hipFree(d_tree.nodes);
            return -1;
        }
        break;
    }
    // 根据数据形式和访问方式的不同执行不同的核函数
    switch(this->df) {
    case df_1D: {
        // 申请 out 数组空间
        cuerrcode = hipMalloc((void**)&dev_out1D, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) {
            hipFree(d_data1D);
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_out1D, 0, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) {
            hipFree(dev_out1D);
            hipFree(d_data1D);
            return -1;
        }
        // 核函数
        // 计算线程数
        int gridsize_1d, blocksize_1d;
        blocksize_1d = this->block_size;
        gridsize_1d = (this->thread_num + blocksize_1d - 1) / blocksize_1d;
        // 每个线程拷贝多少个数据到shared memory
        int copy_num_per_thread = (this->size + this->block_size - 1) / this->block_size;
        if (this->am == am_step) {
            _d1DSharedStepKer<<<gridsize_1d, blocksize_1d, this->size>>>
                             (d_data1D, dev_out1D, this->step, this->am_num, this->size, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                return -3;
            }
        } else if (this->am == am_sequential) {
            _d1DSharedSequentialKer<<<gridsize_1d, blocksize_1d, this->size>>>
                                   (d_data1D, dev_out1D, this->am_num, this->size, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                return -3;
            }
        } else {
            // device 端访问下标数据
            int *dev_am_data;
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                return -3;
            }
            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(float) * this->size, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                hipFree(dev_am_data);
                return -3;
            }

            _d1DSharedCommonKer<<<gridsize_1d, blocksize_1d, this->size>>>
                               (d_data1D, dev_out1D, this->am_num, this->size, dev_am_data, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out1D);
                hipFree(dev_am_data);
                return -3;
            }
        }
        break;
    }
    case df_2D: 
    {
        // 计算二维数组的大小，这里重新计算的是为了方式只测试二维数组时，成员变量size
        // 没有及时更新。
        unsigned int size_2D = d_data2D.cols * d_data2D.rows;

        // 申请 out 数组空间
        cuerrcode = hipMalloc((void**)&dev_out2D, sizeof(DATA_TYPE) * size_2D);
        if (cuerrcode != hipSuccess) {
            hipFree(d_data2D.data);
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_out2D, 0, sizeof(DATA_TYPE) * size_2D);
        if (cuerrcode != hipSuccess) {
            hipFree(dev_out2D);
            hipFree(d_data2D.data);
            return -1;
        }
        // 每个线程拷贝多少个数据到shared memory
        int copy_num_per_thread = (size_2D + this->block_size - 1) / this->block_size;
        // 计算线程数，设置二维网格
        // 默认 block.x为 32
        dim3 gridsize, blocksize;
        blocksize.x = DEF_BLOCK_X; // = 32
        blocksize.y = this->block_size / blocksize.x;
        gridsize.x = (d_data2D.cols + blocksize.x - 1) / blocksize.x;
        gridsize.y = (d_data2D.rows + blocksize.y - 1) / blocksize.y;
        if (this->am == am_sequential) {
            _d2DSharedSequentialKer<<<gridsize, blocksize, size_2D>>>(d_data2D, dev_out2D, this->am_num, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data2D.data);
                hipFree(dev_out2D);
                return -3;
            }
        } else if (this->am == am_step) {
            _d2DSharedStepKer<<<gridsize, blocksize, size_2D>>>(d_data2D, dev_out2D, this->step, this->am_num, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(d_data2D.data);
                hipFree(dev_out2D);
                return -3;
            }
        } else {

            // 根据不同访问方式在核函数外生成访问下标
            // P.S. 虽然可以用hiprand在核函数外生成一些分布，但am_geometric分布和am_exponential分布无法生成，
            // 为了考虑性能测试统一，不使用hiprand。
            // 先在 host 端生成下标分布数据，然后再拷贝到 device 端。
            // device 端访问下标数据
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(d_data1D);
                hipFree(dev_out2D);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * size_2D, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(d_data2D.data);
                hipFree(dev_out2D);
                return -3;
            }

            _d2DSharedCommonKer<<<gridsize, blocksize, size_2D>>>(d_data2D, dev_out2D, this->am_num, dev_am_data, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(d_data2D.data);
                hipFree(dev_out2D);
                return -3;
            }
        }
        break;
    }
    case df_tree: 
    {
        cuerrcode = hipMalloc((void**)&dev_outTree, sizeof(DATA_TYPE) * this->tree->num);
        if (cuerrcode != hipSuccess) {
            hipFree(d_tree.nodes);
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_outTree, 0, sizeof(DATA_TYPE) * this->tree->num);
        if (cuerrcode != hipSuccess) {
            hipFree(d_tree.nodes);
            hipFree(dev_outTree);
            return -1;
        }
        int gridsize_1d, blocksize_1d;
        blocksize_1d = this->block_size;
        gridsize_1d = (this->thread_num + blocksize_1d - 1) / blocksize_1d;
        // 每个线程拷贝多少个数据到shared memory
        int copy_num_per_thread = (this->size + this->block_size - 1) / this->block_size;

        // 核函数
        if (this->am == am_sequential) {
            _treeSharedSequentialKer<<<gridsize_1d, blocksize_1d, this->size>>>(d_tree, this->am_num, dev_outTree, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                return -3;
            }
        } else if (this->am == am_step) {
            _treeSharedStepKer<<<gridsize_1d, blocksize_1d, this->size>>>(d_tree, this->step, this->am_num, dev_outTree, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                return -3;
            }
        } else {
            // device 端访问下标数据
            int *dev_am_data;
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * this->size, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                hipFree(dev_am_data);
                return -3;
            }

            _treeSharedCommonKer<<<gridsize_1d, blocksize_1d, this->size>>>(d_tree, this->am_num, dev_outTree, dev_am_data, copy_num_per_thread);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                hipFree(d_tree.nodes);
                hipFree(dev_am_data);
                return -3;
            }
        }
        break;
    }
    }

    if (d_data1D == NULL)
        hipFree(d_data1D);
    if (d_data2D.data == NULL)
        hipFree(d_data2D.data);
    if (d_tree.nodes == NULL)
        hipFree(d_tree.nodes);

    if (dev_out1D == NULL)
        hipFree(dev_out1D);
    if (dev_out2D == NULL)
        hipFree(dev_out2D);
    if (dev_outTree == NULL)
        hipFree(dev_outTree);
    if (dev_am_data == NULL)
        hipFree(dev_am_data);

    return 0;
}

int Case::constant_run()
{
    // 错误代码
    hipError_t cuerrcode;

    // 根据内部数据分布，申请空间并拷贝数据
    switch (this->df)
    {
    case df_1D:
        // 数据拷贝
        // (constant_lut, lut, sizeof(unsigned char) * 256);
        cuerrcode = hipMemcpyToSymbol(HIP_SYMBOL(constant_data1D), this->data1D, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) {
            // 数据拷贝出错，返回错误代码前释放申请的空间
            // free(this->data1D);
            return -1;
        }
        break;
    case df_2D:
        // 错误情况判断
        if (this->c == -1 || this->r == -1)
            return -2;
        // 虽然这里的赋值看似无用，但可增强代码健壮性
        data2D.cols = this->c;
        data2D.rows = this->r;

        cuerrcode = hipMemcpyToSymbol(HIP_SYMBOL(constant_data2D), this->data2D.data, sizeof(DATA_TYPE) * this->c * this->r);
        if (cuerrcode != hipSuccess) {
            // 数据拷贝出错，返回错误代码前释放申请的空间
            // free(this->data2D.data);
            return -1;
        }
        break;
    case df_tree:

        cuerrcode = hipMemcpyToSymbol(HIP_SYMBOL(constant_treeNodes), this->tree->nodes, sizeof(Node) * this->tree->num);
        if (cuerrcode != hipSuccess) {
            // hipFree(this->tree->nodes);
            return -1;
        }
        break;
    }
    // 根据数据形式和访问方式的不同执行不同的核函数
    switch(this->df) {
    case df_1D:
    {
        DATA_TYPE *dev_out1D;
        // 申请 out 数组空间
        cuerrcode = hipMalloc((void**)&dev_out1D, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) {
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_out1D, 0, sizeof(DATA_TYPE) * this->size);
        if (cuerrcode != hipSuccess) {
            hipFree(dev_out1D);
            return -1;
        }
        // 核函数
        // 计算线程数
        int gridsize_1d, blocksize_1d;
        blocksize_1d = this->block_size;
        gridsize_1d = (this->thread_num + blocksize_1d - 1) / blocksize_1d;
        if (this->am == am_step) {
            _d1DConstantStepKer<<<gridsize_1d, blocksize_1d>>>(dev_out1D, this->step, this->am_num, this->size);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_out1D);
                return -3;
            }
        } else if (this->am == am_sequential) {
            _d1DConstantSequentialKer<<<gridsize_1d, blocksize_1d>>>(dev_out1D, this->am_num, this->size);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_out1D);
                return -3;
            }
        } else {
            // device 端访问下标数据
            int *dev_am_data;
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_out1D);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * this->size, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_out1D);
                hipFree(dev_am_data);
                return -3;
            }
            // hiprandDestroyGenerator(gen);

            _d1DConstantCommonKer<<<gridsize_1d, blocksize_1d>>>(dev_out1D, this->am_num, dev_am_data, this->size);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_out1D);
                hipFree(dev_am_data);
                return -3;
            }
        }
        break;
    }
    case df_2D:
    {
        DATA_TYPE *dev_out2D;
        // 计算二维数组的大小，这里重新计算的是为了方式只测试二维数组时，成员变量size
        // 没有及时更新。
        unsigned int size_2D = this->c * this->r;
        // 申请 out 数组空间
        cuerrcode = hipMalloc((void**)&dev_out2D, sizeof(DATA_TYPE) * size_2D);
        if (cuerrcode != hipSuccess) {
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_out2D, 0, sizeof(DATA_TYPE) * size_2D);
        if (cuerrcode != hipSuccess) {
            hipFree(dev_out2D);
            return -1;
        }
        // 计算线程数，设置二维网格
        // 默认 block.x为 32
        dim3 gridsize, blocksize;
        blocksize.x = DEF_BLOCK_X; // = 32
        blocksize.y = this->block_size / blocksize.x;
        gridsize.x = (this->c + blocksize.x - 1) / blocksize.x;
        gridsize.y = (this->r + blocksize.y - 1) / blocksize.y;
        if (this->am == am_sequential) {
            _d2DConstantSequentialKer<<<gridsize, blocksize>>>(dev_out2D, this->am_num, this->data2D.cols, this->data2D.rows);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_out2D);
                return -3;
            }
        } else if (this->am == am_step) {
            _d2DConstantStepKer<<<gridsize, blocksize>>>(dev_out2D, this->step, this->am_num, this->data2D.cols, this->data2D.rows);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_out2D);
                return -3;
            }
        } else {

            // 根据不同访问方式在核函数外生成访问下标
            // P.S. 虽然可以用hiprand在核函数外生成一些分布，但am_geometric分布和am_exponential分布无法生成，
            // 为了考虑性能测试统一，不使用hiprand。
            // 先在 host 端生成下标分布数据，然后再拷贝到 device 端。
            // device 端访问下标数据
            int *dev_am_data;
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_out2D);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * size_2D, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(dev_out2D);
                return -3;
            }

            _d2DConstantCommonKer<<<gridsize, blocksize>>>(dev_out2D, this->am_num, dev_am_data, this->data2D.cols, this->data2D.rows);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(dev_out2D);
                return -3;
            }
        }
        break;
    }
    case df_tree:
    {
        DATA_TYPE *dev_outTree;
        cuerrcode = hipMalloc((void**)&dev_outTree, sizeof(DATA_TYPE) * this->tree->num);
        if (cuerrcode != hipSuccess) {
            return -1;
        }
        cuerrcode = hipMemset((void*)dev_outTree, 0, sizeof(DATA_TYPE) * this->tree->num);
        if (cuerrcode != hipSuccess) {
            hipFree(dev_outTree);
            return -1;
        }
        int gridsize_1d, blocksize_1d;
        blocksize_1d = this->block_size;
        gridsize_1d = (this->thread_num + blocksize_1d - 1) / blocksize_1d;
        // 核函数
        if (this->am == am_sequential) {
            _treeConstantSequentialKer<<<gridsize_1d, blocksize_1d>>>(this->size, this->am_num, dev_outTree);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                return -3;
            }
        } else if (this->am == am_step) {
            _treeConstantStepKer<<<gridsize_1d, blocksize_1d>>>(this->size, this->step, this->am_num, dev_outTree);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_outTree);
                return -3;
            }
        } else {
            // device 端访问下标数据
            int *dev_am_data;
            cuerrcode = hipMalloc((void**)&dev_am_data, sizeof(int) * this->size);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_outTree);
                return -3;
            }

            cuerrcode = hipMemcpy(dev_am_data, host_am_data, sizeof(int) * this->size, hipMemcpyHostToDevice);
            if (cuerrcode != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(dev_outTree);
                return -3;
            }

            _treeConstantCommonKer<<<gridsize_1d, blocksize_1d>>>(this->size, this->am_num, dev_outTree, dev_am_data);
            if (hipGetLastError() != hipSuccess) {
                hipFree(dev_am_data);
                hipFree(dev_outTree);
                return -3;
            }
        }
        break;
    }
    }
    return 0;
}
